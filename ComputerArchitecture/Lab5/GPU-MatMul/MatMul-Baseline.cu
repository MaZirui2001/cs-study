#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace std;

#define TILE_WIDTH 32
// CUDA核函数：矩阵乘法
__global__ void gemm_baseline(float *A, float *B, float *C, int N)
{
    int threadId = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
	if (threadId >= N * N){
		return;
    }
	int row = threadId / N;
	int col = threadId % N;

	C[threadId] = 0;
	for (size_t i = 0; i < N; i++){
		C[threadId] += A[row * N + i] * B[i * N + col];
	}
}
// void gemm_baseline_CPU(const float *A, const float *B, float *C) {
//    for (int i = 0; i < N; i++) {
//        for (int k = 0; k < N; k++) {
//            for (int j = 0; j < N; j++) {
//                C[i * N + j] += A[i * N + k] * B[k * N + j];
//            }
//        }
//    }
// }

void gemm_verify(float *A, float *B, float *C, int N) {
    auto *D = (float *)malloc(N * N * sizeof(float));
    memset (D, 0, N * N * sizeof(float));
    
    for (int i = 0; i < N; i++) {
        for (int k = 0; k < N; k++) {
            for(int j = 0; j < N; j++) {
                D[i * N + j] += A[i * N + k] * B[k * N + j];
            }
        }
    }
    for (int i = 0; i < N * N; i++) {
        if (abs(C[i] - D[i]) > 1e-1 && abs(C[i] - D[i] / D[i]) > 1e-2) {
            cout << "Wrong!" << endl;
            return;
        }
    }
    cout << "Correct!" << endl;
    free(D);
}

int main()
{
    int N = 1 << 10; // 矩阵宽度
    int size = N * N * sizeof(float);

    // 分配主机内存
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);
    uniform_real_distribution<double> u(-10, 10);
    default_random_engine e(time(nullptr));
    // 初始化矩阵
    for (int i = 0; i < N * N; i++) {
        h_A[i] = u(e);
        h_B[i] = u(e);
        h_C[i] = 0.0f;
    }

    // 分配设备内存
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // 将主机内存数据复制到设备内存
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // 定义CUDA核函数的块大小和线程大小
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
    dim3 gridDim((N + TILE_WIDTH - 1) / TILE_WIDTH, (N + TILE_WIDTH - 1) / TILE_WIDTH);

    // 调用CUDA核函数
    // 高精度计时
    auto start = chrono::system_clock::now();
    for(int i = 0; i < 10; i++)
        gemm_baseline<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);
    auto end = chrono::system_clock::now();
    

    // 将设备内存数据复制到主机内存
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    auto duration = chrono::duration_cast<chrono::microseconds>(end - start);
    cout << "elapsed time: " << duration.count() << "micros\n";

    // 验证结果
    gemm_verify(h_A, h_B, h_C, N);

    // 释放内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}