#include <iostream>
#include <hip/hip_runtime.h>

#include <random>
#include <cstdint>
using namespace std;

//#define CHECK
#define TILE_WIDTH 32
// CUDA核函数：矩阵乘法
__global__ void gemm_baseline(const float* A, const float* B, float* C, int N)
{
    uint32_t threadId = (blockIdx.y * blockDim.y + threadIdx.y) * gridDim.x * blockDim.x + blockIdx.x * blockDim.x + threadIdx.x;
    if (threadId >= N * N) {
        return;
    }
    uint32_t row = threadId / N;
    uint32_t col = threadId % N;

    C[threadId] = 0;
    for (size_t i = 0; i < N; i++) {
        C[threadId] += A[row * N + i] * B[i * N + col];
    }
}
#ifdef CHECK
void gemm_verify(const float* A, const float* B, float* C, int N) {
    auto* D = (float*)malloc(N * N * sizeof(float));
    memset(D, 0, N * N * sizeof(float));

    for (int i = 0; i < N; i++) {
        for (int k = 0; k < N; k++) {
            for (int j = 0; j < N; j++) {
                D[i * N + j] += A[i * N + k] * B[k * N + j];
            }
        }
    }
    for (int i = 0; i < N * N; i++) {
        if (abs(C[i] - D[i]) > 1e-1 && abs(C[i] - D[i] / D[i]) > 1e-2) {
            cout << "Wrong!" << endl;
            return;
        }
    }
    cout << "Correct!" << endl;
    free(D);
}
#endif
int main()
{
    uint32_t N = 1 << 10; // 矩阵宽度
    uint32_t size = N * N * sizeof(float);

    // 分配主机内存
    auto* h_A = (float*)malloc(size);
    auto* h_B = (float*)malloc(size);
    auto* h_C = (float*)malloc(size);
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    uniform_real_distribution<double> u(-10, 10);
    default_random_engine e(time(nullptr));
    // 初始化矩阵
    for (int i = 0; i < N * N; i++) {
        h_A[i] = (float)u(e);
        h_B[i] = (float)u(e);
        h_C[i] = 0.0f;
    }
    // 分配设备内存
    float* d_A, * d_B, * d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // 将主机内存数据复制到设备内存
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // 定义CUDA核函数的块大小和线程大小
    dim3 block_dim(TILE_WIDTH, TILE_WIDTH);
    dim3 grid_dim((N + TILE_WIDTH - 1) / TILE_WIDTH, (N + TILE_WIDTH - 1) / TILE_WIDTH);

    // 调用CUDA核函数
    // 高精度计时
    hipEventRecord(start, nullptr);
    gemm_baseline <<<grid_dim, block_dim >>> (d_A, d_B, d_C, N);
    hipEventRecord(stop, nullptr);
    hipEventSynchronize(stop);

    // 将设备内存数据复制到主机内存
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    float duration;
    hipEventElapsedTime(&duration, start, stop);
    cout << "elapsed time: " << duration << "micros\n";

    // 验证结果
#ifdef CHECK
    gemm_verify(h_A, h_B, h_C, N);
#endif

    // 释放内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}