#include <iostream>
#include <hip/hip_runtime.h>
#include <>
#include <random>
using namespace std;
// Device Name : NVIDIA GeForce GTX 1650 Ti
// totalGlobalMem : 4294705152
// sharedMemPerBlock 49152
// regsPerBlock : 65536
// warpSize : 32
// memPitch : 2147483647
// maxThreadsPerBlock 1024
// maxThreadsDim[0 - 2] : 1024 1024 64
// maxGridSize[0 - 2] 2147483647 65535 65535
// totalConstMem : 65536
// major.minor : 7.5
// clockRate : 1485000
// textureAlignment : 512
// deviceOverlap : 1
// multiProcessorCount : 16

void printDeviceProp(const hipDeviceProp_t& prop)
{
    cout << "Device Name : " << prop.name << "\n";
    cout << "totalGlobalMem : " << prop.totalGlobalMem << "\n";
    cout << "sharedMemPerBlock " << prop.sharedMemPerBlock << "\n";
    cout << "regsPerBlock : " << prop.regsPerBlock << "\n";
    cout << "warpSize :" << prop.warpSize << "\n";
    cout << "memPitch : " << prop.memPitch << "\n";
    cout << "maxThreadsPerBlock " << prop.maxThreadsPerBlock << "\n";
    cout << "maxThreadsDim[0 - 2] : " << prop.maxThreadsDim[0] << " " << prop.maxThreadsDim[1] << " " << prop.maxThreadsDim[2] << "\n";
    cout << "maxGridSize[0 - 2] " << prop.maxGridSize[0] << " " << prop.maxGridSize[1] << " " << prop.maxGridSize[2] << "\n";
    cout << "totalConstMem : " << prop.totalConstMem << "\n";
    cout << "major.minor : " << prop.major << "." << prop.minor << "\n";
    cout << "clockRate : " << prop.clockRate << "\n";
    cout << "textureAlignment :" << prop.textureAlignment << "\n";
    cout << "deviceOverlap : " << prop.deviceOverlap << "\n";
    cout << "multiProcessorCount : " << prop.multiProcessorCount << "\n";
}
int main() {
    hipDeviceProp_t prop{};
    hipGetDeviceProperties(&prop, 0);
    printDeviceProp(prop);
    return 0;
}
