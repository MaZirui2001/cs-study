#include <iostream>
#include <hip/hip_runtime.h>

#include <random>
#include <cstdint>
using namespace std;
//#define CHECK
#define TILE_WIDTH 32
#define BLOCK_SIZE TILE_WIDTH
// CUDA核函数：矩阵乘法
__global__ void gemm_block(const float* A, const float* B, float* C, int N)
{
    if ((blockIdx.y * blockDim.y + threadIdx.y) * N + blockIdx.x * blockDim.x + threadIdx.x >= N * N) {
        return;
    }

    uint32_t const begin_a = blockIdx.y * blockDim.y * N;
    uint32_t const end_a = begin_a + N - 1;
    uint32_t const step_a = blockDim.x;

    uint32_t const begin_b = blockIdx.x * blockDim.x;
    uint32_t const step_b = blockDim.y * N;

    float result_temp = 0.0f;
    uint32_t i_a = begin_a;
    uint32_t i_b = begin_b;
    for (; i_a < end_a; i_a += step_a, i_b += step_b)  {
        // shared memory
        __shared__ float ASub[BLOCK_SIZE][BLOCK_SIZE];
        __shared__ float BSub[BLOCK_SIZE][BLOCK_SIZE];

        // copy data to shared memory
        ASub[threadIdx.y][threadIdx.x] = A[i_a + threadIdx.y * N + threadIdx.x];
        BSub[threadIdx.y][threadIdx.x] = B[i_b + threadIdx.y * N + threadIdx.x];

        __syncthreads();

        for (int i = 0; i < BLOCK_SIZE; i++) {
            result_temp += ASub[threadIdx.y][i] * BSub[i][threadIdx.x];
        }

        __syncthreads();
    }

    uint32_t begin_result = blockIdx.y * blockDim.y * N + begin_b;
    C[begin_result + threadIdx.y * N + threadIdx.x] = result_temp;
}
#ifdef CHECK
void gemm_verify(const float* A, const float* B, const float* C, int N) {
    auto* D = (float*)malloc(N * N * sizeof(float));
    memset(D, 0, N * N * sizeof(float));

    for (int i = 0; i < N; i++) {
        for (int k = 0; k < N; k++) {
            for (int j = 0; j < N; j++) {
                D[i * N + j] += A[i * N + k] * B[k * N + j];
            }
        }
    }
    for (int i = 0; i < N * N; i++) {
        if (abs(C[i] - D[i]) > 1e-1 && abs(C[i] - D[i] / D[i]) > 1e-2) {
            cout << "Wrong!" << endl;
            return;
        }
    }
    cout << "Correct!" << endl;
    free(D);
}
#endif
int main()
{
    int N = 1 << 11; // 矩阵宽度
    uint32_t size = N * N * sizeof(float);

    // 分配主机内存
    auto * h_A = (float*)malloc(size);
    auto * h_B = (float*)malloc(size);
    auto * h_C = (float*)malloc(size);


    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    uniform_real_distribution<double> u(-10, 10);
    default_random_engine e(time(nullptr));

    // 初始化矩阵
    for (int i = 0; i < N * N; i++) {
        h_A[i] = (float)u(e);
        h_B[i] = (float)u(e);
        h_C[i] = 0.0f;
    }
    // 分配设备内存
    float* d_A, * d_B, * d_C;
    hipMalloc((void**)&d_A, size);
    hipMalloc((void**)&d_B, size);
    hipMalloc((void**)&d_C, size);

    // 将主机内存数据复制到设备内存
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // 定义CUDA核函数的块大小和线程大小
    dim3 block_dim(TILE_WIDTH, TILE_WIDTH);
    dim3 grid_dim((N + TILE_WIDTH - 1) / TILE_WIDTH, (N + TILE_WIDTH - 1) / TILE_WIDTH);

    // 调用CUDA核函数
    // 高精度计时
    hipEventRecord(start, nullptr);
    gemm_block <<<grid_dim, block_dim >>> (d_A, d_B, d_C, N);
    hipEventRecord(stop, nullptr);
    hipEventSynchronize(stop);


    // 将设备内存数据复制到主机内存
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    float duration;
    hipEventElapsedTime(&duration, start, stop);
    cout << "elapsed time: " << duration << "micros\n";

    // 验证结果
#ifdef CHECK
    gemm_verify(h_A, h_B, h_C, N);
#endif
    // 释放内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}