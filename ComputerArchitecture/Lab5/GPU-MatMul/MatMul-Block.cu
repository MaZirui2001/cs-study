#include <bits/stdc++.h>
#include <hip/hip_runtime.h>
using namespace std;

#define TILE_WIDTH 32
#define BLOCK_SIZE 32
// CUDA核函数：矩阵乘法
__global__ void gemm_baseline(float *A, float *B, float *C, int N)
{
// thread location

	if ((blockIdx.y * blockDim.y + threadIdx.y) * N + blockIdx.x * blockDim.x + threadIdx.x >= N * N){
		return;
	}

	// blockDim.x == blockDim.y == BLOCK_SIZE here
	const int begin_a = blockIdx.y * blockDim.y * N;
	const int end_a = begin_a + N - 1;
	const int step_a = blockDim.x;

	const int begin_b = blockIdx.x * blockDim.x;
	const int step_b = blockDim.y * N;

	float result_temp = 0.0f;
    int index_a = begin_a;
    int index_b = begin_b;
	for (; index_a < end_a; index_a += step_a, index_b += step_b)
	{
		// shared memory
		__shared__ float SubMat_A[BLOCK_SIZE][BLOCK_SIZE];
		__shared__ float SubMat_B[BLOCK_SIZE][BLOCK_SIZE];

		// copy data to shared memory
		SubMat_A[threadIdx.y][threadIdx.x] = A[index_a + threadIdx.y * N + threadIdx.x];
		SubMat_B[threadIdx.y][threadIdx.x] = B[index_b + threadIdx.y * N + threadIdx.x];

		__syncthreads();

		for (int i = 0; i < BLOCK_SIZE; i++)
		{
			result_temp += SubMat_A[threadIdx.y][i] * SubMat_B[i][threadIdx.x];
		}

		__syncthreads();
	}

	int begin_result = blockIdx.y * blockDim.y * N + begin_b;
	C[begin_result + threadIdx.y * N + threadIdx.x] = result_temp;
}

void gemm_verify(float *A, float *B, float *C, int N) {
    auto *D = (float *)malloc(N * N * sizeof(float));
    memset (D, 0, N * N * sizeof(float));
    
    for (int i = 0; i < N; i++) {
        for (int k = 0; k < N; k++) {
            for(int j = 0; j < N; j++) {
                D[i * N + j] += A[i * N + k] * B[k * N + j];
            }
        }
    }
    for (int i = 0; i < N * N; i++) {
        if (abs(C[i] - D[i]) > 1e-1 && abs(C[i] - D[i] / D[i]) > 1e-2) {
            cout << "Wrong!" << endl;
            return;
        }
    }
    cout << "Correct!" << endl;
    free(D);
}

int main()
{
    int N = 1 << 10; // 矩阵宽度
    int size = N * N * sizeof(float);

    // 分配主机内存
    float *h_A = (float *)malloc(size);
    float *h_B = (float *)malloc(size);
    float *h_C = (float *)malloc(size);
    uniform_real_distribution<double> u(-10, 10);
    default_random_engine e(time(nullptr));
    // 初始化矩阵
    for (int i = 0; i < N * N; i++) {
        h_A[i] = u(e);
        h_B[i] = u(e);
        h_C[i] = 0.0f;
    }

    // 分配设备内存
    float *d_A, *d_B, *d_C;
    hipMalloc((void **)&d_A, size);
    hipMalloc((void **)&d_B, size);
    hipMalloc((void **)&d_C, size);

    // 将主机内存数据复制到设备内存
    hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
    hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    // 定义CUDA核函数的块大小和线程大小
    dim3 blockDim(TILE_WIDTH, TILE_WIDTH);
    dim3 gridDim((N + TILE_WIDTH - 1) / TILE_WIDTH, (N + TILE_WIDTH - 1) / TILE_WIDTH);

    // 调用CUDA核函数
    // 高精度计时
    auto start = chrono::system_clock::now();
    for(int i = 0; i < 10; i++)
        gemm_baseline<<<gridDim, blockDim>>>(d_A, d_B, d_C, N);
    auto end = chrono::system_clock::now();
    

    // 将设备内存数据复制到主机内存
    hipMemcpy(h_C, d_C, size, hipMemcpyDeviceToHost);
    auto duration = chrono::duration_cast<chrono::microseconds>(end - start);
    cout << "elapsed time: " << duration.count() << "micros\n";

    // 验证结果
    gemm_verify(h_A, h_B, h_C, N);

    // 释放内存
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);
    free(h_A);
    free(h_B);
    free(h_C);

    return 0;
}